#include <stdio.h>
#include <locale.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


long getmax(long *, long);



__global__ void getMaxNum( long *in, long size, long blocks_d, long *out) {

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x *blockDim.x + threadIdx.x;

    // each thread loads one element from global to shared mem
    long x = LONG_MIN; 
    if(i < size) 
        x = in[i]; 
    out[blockIdx.x*blocks_d+tid] = x;

    __syncthreads();


    for (unsigned int s=blockDim.x/2; s>0; s = s/2){
        if (tid < s){

            __syncthreads();

            if (out[blockIdx.x * blocks_d+tid] < out[blockIdx.x * blocks_d+tid+s]){
                out[blockIdx.x * blocks_d+tid] = out[blockIdx.x * blocks_d+tid+s];
            }
           
        }
        __syncthreads();
    }

    __syncthreads();



}


int main(int argc, char *argv[])
{
    long size = 0;  // The size of the array
    long i;  // loop index
    long max;
    long * numbers; //pointer to the array

    if(argc !=2)
    {
        printf("usage: maxseq num\n");
        printf("num = size of the array\n");
        exit(1);
    }

    size = atol(argv[1]);

    numbers = (long *)malloc(size * sizeof(long));
    if( !numbers )
    {
        printf("Unable to allocate mem for an array of size %ld\n", size);
        exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++){
        numbers[i] = rand() % size; 
    }

   
    int numsSize = size * sizeof(long);


    // Get the number of threads per block but getting the device's
    // maximum threads per block
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    long THREADS_PER_BLOCK = devProp.maxThreadsPerBlock;

    //Create nums array that we will be sending to the device
    long * nums;

    // Get number of blocks by rounding up the size of the array / threads per block
    // so, the amount of blocks needed for the max threads per block for this device
    long blocks = ((size + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK);
    long numOfThreads;
    if (size > THREADS_PER_BLOCK){
        numOfThreads = THREADS_PER_BLOCK;
    }
    else{
        numOfThreads = size;
    }

    
    //Transfer and copy numbers array from the host to the device
    hipMalloc((void **) &nums, numsSize);

    hipError_t e = hipMemcpy(nums, numbers, numsSize, hipMemcpyHostToDevice);

    
    // Create array that will store the result - sending this from device to host
    long * maxResult;
    long resultSize = blocks * sizeof(long);

    // Transfer maxResult array to device
    hipMalloc((void **) &maxResult, resultSize);
    hipError_t v  = hipGetLastError();

    //launch kernel function
    getMaxNum<<<blocks, numOfThreads>>>(nums, size, blocks, maxResult);

    // Copy the array from the device to the host so we can get result
    hipError_t s = hipMemcpy(numbers, maxResult, resultSize, hipMemcpyDeviceToHost);

    long l;
    max = numbers[0];

    for(l = 1; l < blocks; l++){
        if(numbers[l] > max){
            max = numbers[l];
        }
    }

    printf("The maximum number in the array is %'ld \n", max);


    hipFree(nums);
    hipFree(maxResult);
    free(numbers);
    hipDeviceReset();
    exit(0);
}
